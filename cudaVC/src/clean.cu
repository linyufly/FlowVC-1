
# include "clean.h"

void tempclean(Point temp) {

	free(temp.x);
	free(temp.y);
	free(temp.z);
	free(temp.ElementIndex);
	free(temp.Start_time);
	free(temp.Stop_time);
	free(temp.LeftDomain); 


}

void clean_gs(void) {

	hipFree(MeshNodeArray_device.x);
	hipFree(MeshNodeArray_device.y);
	hipFree(MeshNodeArray_device.z);

	if (Memory_Usage_GS == Use_Pinned_Memory) {
	
		hipHostFree(MeshNodeArray.x);
		hipHostFree(MeshNodeArray.y);
		hipHostFree(MeshNodeArray.z);
	} else {
		
		free(MeshNodeArray.x);
		free(MeshNodeArray.y);
		free(MeshNodeArray.z);
	
	}
	
}


void clean_all(void) {


		
		// Release all memory allocated

		if (Memory_Usage_GS == Use_Pinned_Memory) {

			hipHostFree(MeshElementArray.Node1);
			hipHostFree(MeshElementArray.Node2);
			hipHostFree(MeshElementArray.Node3);
			hipHostFree(MeshElementArray.Node4);

		//	hipHostFree(MeshNodeArray.x);
		//	hipHostFree(MeshNodeArray.y);
		//	hipHostFree(MeshNodeArray.z);
			
			hipHostFree(MeshNodeArray_double.x);
			hipHostFree(MeshNodeArray_double.y);
			hipHostFree(MeshNodeArray_double.z);

		} else {

			free(MeshElementArray.Node1);
			free(MeshElementArray.Node2);
			free(MeshElementArray.Node3);
			free(MeshElementArray.Node4);

		//	free(MeshNodeArray.x);
		//	free(MeshNodeArray.y);
		//	free(MeshNodeArray.z);
			
			free(MeshNodeArray_double.x);
			free(MeshNodeArray_double.y);
			free(MeshNodeArray_double.z);
			
			
		}
		
		hipFree(MeshElementArray_device.Node1);
		hipFree(MeshElementArray_device.Node2);
		hipFree(MeshElementArray_device.Node3);
		hipFree(MeshElementArray_device.Node4);

//		hipFree(MeshNodeArray_device.x);
//		hipFree(MeshNodeArray_device.y);
//		hipFree(MeshNodeArray_device.z);
		
		hipFree(MeshNodeArray_double_device.x);
		hipFree(MeshNodeArray_double_device.y);
		hipFree(MeshNodeArray_double_device.z);




		free(MeshElementArray.Neighborindex1);
		free(MeshElementArray.Neighborindex2);
		free(MeshElementArray.Neighborindex3);
		free(MeshElementArray.Neighborindex4);
		

		// Cleaning Velocity variables
		
		if (Memory_Usage_Tracer == Use_Pinned_Memory) {
	
			hipHostFree(velocity.u0);
			hipHostFree(velocity.v0);
			hipHostFree(velocity.w0);
			hipHostFree(velocity.u1);
			hipHostFree(velocity.v1);
			hipHostFree(velocity.w1);
			hipHostFree(velocity.time0);
			hipHostFree(velocity.time1);
			
			hipHostFree(Tracer.x);
			hipHostFree(Tracer.y);
			
			//if(Dimensions == 3)
				hipHostFree(Tracer.z);
				
			hipHostFree(Tracer.ElementIndex);
			hipHostFree(Tracer.LeftDomain);
			hipHostFree(Tracer.Start_time);
			hipHostFree(Tracer.Stop_time);
			if (Trace_ReleaseStrategy == 1) {
				hipHostFree(Tracer1.x);
				hipHostFree(Tracer1.y);
			
				//if(Dimensions == 3)
					hipHostFree(Tracer1.z);
				
				hipHostFree(Tracer1.ElementIndex);
				hipHostFree(Tracer1.LeftDomain);
				hipHostFree(Tracer1.Start_time);
				hipHostFree(Tracer1.Stop_time);
			
				hipHostFree(index1);
				hipHostFree(Tracer.Status);
			}
			hipHostFree(DataTime1);
			hipHostFree(Output_time);
			hipHostFree(Launch_time);
		
		} else {
			free(velocity.u0);
			free(velocity.v0);
			free(velocity.w0);
			free(velocity.u1);
			free(velocity.v1);
			free(velocity.w1);
			free(velocity.time0);
			free(velocity.time1);
			
			free(Tracer.x);
			Tracer.x = NULL;
			free(Tracer.y);
			Tracer.y = NULL;
			
		
			free(Tracer.z);
			Tracer.z = NULL;
		
			free(Tracer.ElementIndex);
			Tracer.ElementIndex = NULL;
			free(Tracer.Start_time);
			Tracer.Start_time = NULL;
			free(Tracer.Stop_time);
			Tracer.Stop_time = NULL;
			free(Tracer.LeftDomain);
			Tracer.LeftDomain = NULL;
			
			if (Trace_ReleaseStrategy == 1) {
				free(Tracer1.x);
				Tracer1.x = NULL;
				free(Tracer1.y);
				Tracer1.y = NULL;
			
		
				free(Tracer1.z);
				Tracer1.z = NULL;
		
				free(Tracer1.ElementIndex);
				Tracer1.ElementIndex = NULL;
				free(Tracer1.Start_time);
				Tracer1.Start_time = NULL;
				free(Tracer1.Stop_time);
				Tracer1.Stop_time = NULL;
				free(Tracer1.LeftDomain);
				Tracer1.LeftDomain = NULL;
			
				free(index1);
				index1 = NULL;
				
				free(Tracer.Status);
				Tracer.Status = NULL;
			}
			free(DataTime1);
			free(Output_time);
			free(Launch_time);
		}
		
		hipFree(velocity_dev.u0);
		hipFree(velocity_dev.v0);
		hipFree(velocity_dev.w0);
		hipFree(velocity_dev.u1);
		hipFree(velocity_dev.v1);
		hipFree(velocity_dev.w1);
		

		hipFree(Tracer_dev.x);
		hipFree(Tracer_dev.y);
		hipFree(Tracer_dev.ElementIndex);
		hipFree(Tracer_dev.Start_time);
		hipFree(Tracer_dev.Stop_time);
		
		
		free(x_host);
		free(y_host);
		
		if(Dimensions == 3) {
			hipFree(Tracer_dev.z);
			free(z_host);
		}
		
	
		
		//hipFree(tempx);
		//hipFree(tempy);
		hipFree(posx);
		hipFree(posy);
		
		
		hipFree(xn0);
		hipFree(xn1);
		
		if (Data_MeshType == UNSTRUCTURED) {
		//	hipFree(eid);
			hipFree(r);
			hipFree(s);
			hipFree(t);
		
		}
		if (Dimensions == 3) {
			hipFree(posz);
			hipFree(xn2);
		
		}
		
		hipFree(integrate);
		
		// Remove Temp file containing tracer release information
		if (!Keep_Tempfile) {
			char BinFile[LONGSTRING];
			sprintf(BinFile, "%s%s.bin", Path_Output, Temp_OutFilePrefix);
			if(remove(BinFile))
					fprintf(stderr, "Warning: Could not delete file %s\n", BinFile);
		}

}
